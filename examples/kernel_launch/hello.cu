#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void hello() {
    printf("Hello, World!\n");
}


int main() {
    hello<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}